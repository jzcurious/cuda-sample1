#include "timers.cuh"

namespace detail {
static double time_placeholder = 0;
}

CPUTimer::CPUTimer()
    : _time(detail::time_placeholder) {}

CPUTimer::CPUTimer(double& time)
    : _time(time)
    , _start(clock_t::now()) {}

CPUTimer::~CPUTimer() {
  time_point_t stop = clock_t::now();
  _time += std::chrono::duration<double>(stop - _start).count();
}

GPUTimer::GPUTimer()
    : _time(detail::time_placeholder) {}

GPUTimer::GPUTimer(double& time)
    : _time(time) {
  hipDeviceSynchronize();
  hipEventCreate(&_start);
  hipEventCreate(&_stop);
  hipEventRecord(_start);
}

GPUTimer::~GPUTimer() {
  hipDeviceSynchronize();
  hipEventRecord(_stop);
  hipEventSynchronize(_stop);
  float delta;
  hipEventElapsedTime(&delta, _start, _stop);
  hipEventDestroy(_start);
  hipEventDestroy(_stop);
  _time += static_cast<double>(delta) / 1000;
}
